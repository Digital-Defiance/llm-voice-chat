#include <hip/hip_runtime.h> 


__global__ void add_vectors_kernel(float *a, float *b, float *c) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    c[i] += a[i] + b[i];
}



const size_t SIZE_OF_VEC2 = 2*sizeof(float);

extern "C" {
    void add_vectors_cuda(float *a, float *b, float *c) {
        int numBlocks = 1;
        int numThreadsPerBlock = 2;


        // Allocate memory for arrays d_A, d_B, and d_C on device
        float *d_A, *d_B, *d_C;
    
        hipMalloc(&d_A, SIZE_OF_VEC2);
        hipMalloc(&d_B, SIZE_OF_VEC2);
        hipMalloc(&d_C, SIZE_OF_VEC2);

        // Copy data from host arrays A and B to device arrays d_A and d_B
        hipMemcpy(d_A, a, SIZE_OF_VEC2, hipMemcpyHostToDevice);
        hipMemcpy(d_B, b, SIZE_OF_VEC2, hipMemcpyHostToDevice);


        add_vectors_kernel<<<numBlocks, numThreadsPerBlock>>>(d_A, d_B, d_C);
    
    	hipMemcpy(c, d_C, SIZE_OF_VEC2, hipMemcpyDeviceToHost);

        hipDeviceSynchronize();
    }
}
