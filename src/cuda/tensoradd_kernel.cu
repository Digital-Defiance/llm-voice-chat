#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h> 
#include <torch/extension.h>
#include <hip/hip_runtime.h>


template <typename scalar_t>
__global__ void add_tensors_kernel(scalar_t *a, scalar_t *b, scalar_t *c) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    c[i] += a[i] + b[i];
}

extern "C" {
    void add_tensors_cuda(torch::Tensor *result, torch::Tensor *a, torch::Tensor *b) {
        AT_DISPATCH_FLOATING_TYPES(a->type(), "cuda_add_tensors", ([&] {
            add_tensors_kernel<scalar_t><<<2, 1>>>(
                a->data<scalar_t>(),
                b->data<scalar_t>(),
                result->data<scalar_t>()
            );
        }));
    }
}
