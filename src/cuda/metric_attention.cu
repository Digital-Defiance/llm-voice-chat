#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h> 
#include <torch/extension.h>
#include <hip/hip_runtime.h>

#include <torch/torch.h>

using namespace torch::autograd;

#define CHECK_CUDA(x) TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA((*x)); CHECK_CONTIGUOUS((*x))

typedef torch::Tensor *TensorPTR;
typedef const int Vec1[];
template<typename scalar_t, size_t D>
using CudaTensorView = torch::PackedTensorAccessor32<scalar_t, D, torch::RestrictPtrTraits>;

using Constant1DLookup = CudaTensorView<size_t, 1>;

using constants_list = std::vector<at::Tensor>;


template <typename scalar_t> 
__global__ void metric_attention_forwards_kernel(
    CudaTensorView<scalar_t, 4> p_bnck,
    Constant1DLookup f_l,
    Constant1DLookup g_l,
    Constant1DLookup f_u,
    Constant1DLookup g_u,
    CudaTensorView<scalar_t, 2> M_nl,
    CudaTensorView<scalar_t, 4> q_bnul
) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Global thread index
    
    int Nb = q_bnul.size(0);
    int b = idx % Nb;
    idx = (idx / Nb);
    
    int Nn = q_bnul.size(1);
    int n = idx % Nn;
    idx = idx / Nn;

    int Nl = q_bnul.size(3);
    int l = idx % Nl;
    idx = idx / Nl;

    int Nl = q_bnul.size(2);
    int u = idx % Nu;

    int fu = f_u[u];
    int gu = g_u[u];

    int fl = f_l[l];
    int gl = g_l[l];

    if (fl == gl and fu == gu){
        q_bnul[b][n][u][l] = M_nl[n][l]*p_bnck[b][n][fu][fl]*p_bnck[b][n][fu][fl];
    } else if (fl == gl and fu != gu) {
        q_bnul[b][n][u][l] = 2*M_nl[n][l]*p_bnck[b][n][fu][fl]*p_bnck[b][n][gu][fl];
    } else if (fl != gl and fu == gu) {
        q_bnul[b][n][u][l] = 2*M_nl[n][l]*p_bnck[b][n][fu][fl]*p_bnck[b][n][fu][gl];
    } else if (fl != gl and fu != gu) {
        q_bnul[b][n][u][l] = 4*M_nl[n][l]*p_bnck[b][n][fu][fl]*p_bnck[b][n][gu][gl];
    }
}


template <typename scalar_t> 
__global__ void metric_attention_backwards_kernel(
    CudaTensorView<scalar_t, 4> p_bnck,
    Vec1 f_l, Vec1 g_l, Vec1 f_u, Vec1 g_u,
    CudaTensorView<scalar_t, 2> M_nl,
    CudaTensorView<scalar_t, 4> q_bnul
) {
    /// TODO metric_attention_backwards_kernel
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    grad_input_bcd[i] = grad_output_bcd[i]*metric_1nkk[i]*metric_1nkk[i];
    grad_metric_1nkk[i] = grad_output_bcd[i]*2*input_bcd[i]*metric_1nkk[i];
}


class MetricTensorAttention : public Function<MetricTensorAttention> {
    public:
        static variable_list forward(
            AutogradContext *ctx,
            Variable p_bnck,
            Variable M_nl,
            constants_list constants
        ) {

            const auto device = p_bnck.device();
            auto q_nul = torch::zeros(p_bnck.sizes()).to(device);
    
            auto f_l = constants[0];
            auto g_l = constants[1];
            auto f_u = constants[2];
            auto g_u = constants[3];


            const auto Nb = p_bnck.size(0);
            const auto Nl = M_nl.size(1);
            const auto Nn = M_nl.size(0);
            const auto Nu = f_u.size(0);

            const int total_threads = Nb*Nl*Nu*Nn;
            const int threads_per_block = 1024;
            const int number_of_blocks = (total_threads + threads_per_block - 1) / threads_per_block;
            
            auto q_bnul = torch::zeros((Nb, Nn, Nu, Nl));

            AT_DISPATCH_FLOATING_TYPES(p_bnck.type(), "metric_attention_forwards_kernel", ([&] {
                metric_attention_forwards_kernel<scalar_t><<<number_of_blocks, threads_per_block>>>(
                    p_bnck.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>(),
                    f_l.packed_accessor32<size_t, 1, torch::RestrictPtrTraits>(),
                    g_l.packed_accessor32<size_t, 1, torch::RestrictPtrTraits>(),
                    f_u.packed_accessor32<size_t, 1, torch::RestrictPtrTraits>(),
                    g_u.packed_accessor32<size_t, 1, torch::RestrictPtrTraits>(),
                    M_nl.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                    q_bnul.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>()
                );
            }));

            ctx->save_for_backward({ q_bnul, M_nl, f_l, g_l, f_u, g_u });


            return { q_bnul };
        }

        static tensor_list backward(AutogradContext *ctx, tensor_list grad_outputs) {

            torch::Tensor grad_q_bnul = grad_outputs[0];

            auto saved = ctx->get_saved_variables();
            auto q_bnul = saved[0];
            auto M_nl = saved[1];
            auto f_l = saved[2];
            auto g_l = saved[3];
            auto f_u = saved[4];
            auto g_u = saved[5];

            const auto device = M_nl.device();
            auto grad_p_bnck = torch::zeros_like(p_bnck).to(device);
            auto grad_M_nl  = torch::zeros_like(M_nl).to(device);

            AT_DISPATCH_FLOATING_TYPES(input_bcd.type(), "metric_attention_backwards_kernel", ([&] {
                metric_attention_backwards_kernel<scalar_t><<<2, 1>>>(
                    grad_q_bnul.data<scalar_t>(),

                    p_bnck.data<scalar_t>(),                    
                    
                    M_nl.data<scalar_t>(),
                    grad_p_bnckd.data<scalar_t>(),
                    grad_M_nl.data<scalar_t>()
                );
            }));

            return {grad_p_bnck, grad_M_nl};
  }
};


extern "C" {

    // note: the naming convention relates to
    // the theoretical derivation present in the readme
    void f_metric_tensor_attention(
        TensorPTR *q_1bnu,
        TensorPTR p_bnck,
        TensorPTR M_nl,
        TensorPTR f_l,
        TensorPTR g_l,
        TensorPTR f_u,
        TensorPTR g_u
    ) {

        CHECK_INPUT(p_bnck);
        CHECK_INPUT(*q_1bnu);
        CHECK_INPUT(f_l); CHECK_INPUT(g_l);
        CHECK_INPUT(f_u); CHECK_INPUT(g_u);
        CHECK_INPUT(M_nl);

        constants_list constants = {*f_l, *g_l, *f_u, *g_u};
        auto res = MetricTensorAttention::apply(
                *p_bnck,
                *M_nl,
                constants
        )[0];
        q_1bnu[0] = new torch::Tensor(res);
    }
}

