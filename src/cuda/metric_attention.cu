#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h> 
#include <torch/extension.h>
#include <hip/hip_runtime.h>

#include <torch/torch.h>

using namespace torch::autograd;

#define CHECK_CUDA(x) TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA((*x)); CHECK_CONTIGUOUS((*x))

typedef torch::Tensor *TensorPTR;

template <typename scalar_t> 
__global__ void metric_attention_forwards_kernel(
        scalar_t *input_bcd,
        scalar_t *output_bcd,
        scalar_t *metric_1nkk
) {
    /// TODO metric_attention_forwards_kernel
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    output_bcd[i] = input_bcd[i]*metric_1nkk[i]*metric_1nkk[i];
}


template <typename scalar_t> 
__global__ void metric_attention_backwards_kernel(
        scalar_t *input_bcd,
        scalar_t *metric_1nkk,

        scalar_t *grad_input_bcd,
        scalar_t *grad_metric_1nkk,
    
        scalar_t *grad_output_bcd
) {
    /// TODO metric_attention_backwards_kernel
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    grad_input_bcd[i] = grad_output_bcd[i]*metric_1nkk[i]*metric_1nkk[i];
    grad_metric_1nkk[i] = grad_output_bcd[i]*2*input_bcd[i]*metric_1nkk[i];
}


// Testing phase, this implements y_bi = w_1i*x_bi**2 for now
class MetricTensorAttention : public Function<MetricTensorAttention> {
    public:
        static torch::Tensor
        forward(
            AutogradContext *ctx,
            torch::Tensor input_bcd,
            torch::Tensor metric_1nkk
        ) {
            ctx->save_for_backward({input_bcd, metric_1nkk });

            auto device = input_bcd.device();
            auto output_bcd = torch::zeros(input_bcd.sizes()).to(device);

            AT_DISPATCH_FLOATING_TYPES(input_bcd.type(), "metric_attention_forwards_kernel", ([&] {
                metric_attention_forwards_kernel<scalar_t><<<2, 1>>>(
                    input_bcd.data<scalar_t>(),
                    output_bcd.data<scalar_t>(),
                    metric_1nkk.data<scalar_t>()
                );
            }));
            return output_bcd;
        }

        static tensor_list
        backward(
            AutogradContext *ctx,
            tensor_list grad_outputs
        ) {

            torch::Tensor grad_output_bcd = grad_outputs[0];

            auto saved = ctx->get_saved_variables();
            torch::Tensor input_bcd = saved[0];
            torch::Tensor metric_1nkk = saved[1];

            auto grad_input_bcd = torch::zeros(input_bcd.sizes()).to(input_bcd.device());
            auto grad_metric_1nkk = torch::zeros(metric_1nkk.sizes()).to(metric_1nkk.device());

            AT_DISPATCH_FLOATING_TYPES(input_bcd.type(), "metric_attention_backwards_kernel", ([&] {
                metric_attention_backwards_kernel<scalar_t><<<2, 1>>>(
                    input_bcd.data<scalar_t>(),
                    metric_1nkk.data<scalar_t>(),
                    
                    grad_input_bcd.data<scalar_t>(),
                    grad_metric_1nkk.data<scalar_t>(),
        
                    grad_output_bcd.data<scalar_t>()
                );
            }));

            return {grad_input_bcd, grad_metric_1nkk};
  }
};


extern "C" {
    void f_metric_tensor_attention(TensorPTR *out, TensorPTR input_bcd, TensorPTR metric_1nkk) {

        CHECK_INPUT(input_bcd);
        CHECK_INPUT(metric_1nkk);
        

        // taken from torch sys:
        // auto outputs__ = torch::abs(*self);
        // out__[0] = new torch::Tensor(outputs__);
        
        auto outputs = MetricTensorAttention::apply(
            *input_bcd,
            *metric_1nkk
        );
        out[0] = new torch::Tensor(outputs);
    }
}

